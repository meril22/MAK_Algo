#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

struct Startup{
    int seed = time(nullptr);    
    int threadsPerBlock = 256;
    int datasetSize = 10000;
	int range = 100;
} startup;

struct DataSet{
    float* values;
    int  size;
};


inline int sizeOfDataSet(DataSet data)
{ return sizeof(float)*data.size; }

DataSet generateRandomDataSet(int size){
    DataSet data;
    data.size = size;
    data.values = (float*)malloc(sizeof(float)*data.size);

    for (int i = 0; i < data.size; i++)
        data.values[i] = (float)(rand()%startup.range);

    return data;
}

bool CompareDataSet(DataSet d1, DataSet d2){

    for (int i = 0; i < d1.size; i++)
        if (d1.values[i] != d2.values[i]){
            printf("Dataset is different");
            return false;
        }
        if (d1.size != d2.size) {printf("Datasets are not equal size\n"); return false;};
    	printf("D1 and D2 are equal!");
    	return true;

}

__global__ void DeviceCalculateSM_Global(float* input, int input_size, float* result, int result_size, int sample_size){
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;
	float sum = 0;
    if (id_x < result_size){

        
        for (int i = 0; i < sample_size; i++)
            sum = sum + input[id_x+i];
           sum = sum/sample_size;

        result[id_x] = sum;
    }
}

__global__ void DeviceCalculateSM_Shared(float* input, int input_size, float* result, int result_size, int sample_size){
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;

    if (id_x < input_size){

        extern __shared__ float cache[];

        int cachedSize = sample_size + blockDim.x;

        for (int i = 0; i < cachedSize/blockDim.x+1; i++){
            int cacheId = threadIdx.x+ i*blockDim.x;
            if (cacheId < cachedSize && cacheId+blockDim.x *blockIdx.x < input_size)
                cache[cacheId] = input[cacheId+blockDim.x *blockIdx.x];
        }
        __syncthreads();

        float sum = 0;
        for (int i = 0; i < sample_size; i++){
            if(i + threadIdx.x < cachedSize && i + id_x < input_size)
                sum = sum + cache[i+threadIdx.x];
        }
        sum = sum/sample_size;

        /*store in global memory*/
        if (id_x < result_size)
            result[id_x] = sum;
    }

}

DataSet CalculateSM(DataSet input, int sample_size, bool usesharedmemory){
    if(sample_size == 1 && input.size < 1 && sample_size < 1 && sample_size > input.size) 
	 { 
	 	printf("Error! Invalid Sample Size"); 
	 	exit(-1); 
	 }
    
    int result_size = input.size-sample_size+1;
    DataSet host_result = {(float*)malloc(sizeof(float)*(result_size)), result_size};

    float* device_input, *device_result;


    int threads_needed = host_result.size;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    if (usesharedmemory){
        int shared_memory_allocation_size = sizeof(float)*(startup.threadsPerBlock+sample_size);
        hipEventRecord(start);
        DeviceCalculateSM_Shared<<<threads_needed/ startup.threadsPerBlock + 1, startup.threadsPerBlock, shared_memory_allocation_size>>> (device_input, input.size, device_result, host_result.size, sample_size);
        hipEventRecord(stop);

    }else{
        hipEventRecord(start);
        DeviceCalculateSM_Global<<<threads_needed/ startup.threadsPerBlock + 1, startup.threadsPerBlock>>> (device_input, input.size, device_result, host_result.size, sample_size);
        hipEventRecord(stop);
    }

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if (usesharedmemory) printf("Shared Memory: "); else printf("Global Memory: ");
    printf("Kernel executed in %f milliseconds\n", milliseconds);

    return host_result;
}

void printDataSet(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.6g, ", data.values[i]);
    printf("\n");
}


int main(int argc, char** argv){

    for (int i = 0; i < argc; i++){
        
        if (strcmp(argv[i],  "Range")==0 && i+1 < argc) startup.range = atoi(argv[i+1]);
        if (strcmp(argv[i],  "Seed")==0 && i+1 < argc) startup.seed = atoi(argv[i+1]);
        if (strcmp(argv[i],  "Block threads")==0 && i+1 < argc) startup.threadsPerBlock = atoi(argv[i+1]);
    }

    srand(startup.seed);

    DataSet data = generateRandomDataSet(100);
    printDataSet( data );
    DataSet shared = CalculateSM(data, 2, true);
    DataSet global = CalculateSM(data, 2, false);


    printDataSet( shared );
    printf("\n");
    printDataSet( global );
    printf("\n");


    printf("Each should be %d elements in size\n", global.size);
    CompareDataSet(global, shared);
}
